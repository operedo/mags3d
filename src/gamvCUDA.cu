#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


#define DT float
#define MAX(x,y)  ((x) >= (y) ? (x) : (y))
#define MIN(x,y)  ((x) < (y) ? (x) : (y))

void Check_CUDA_Error(const char *message)
{
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess) {
		fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
		exit(-1);
	}
}

__device__ void computeVariogram(int i, int  j,const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    float *sh_np,float *sh_dis,float *sh_tm,float *sh_hm,float *sh_gam,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr){

//    int half_nd = nd/2;
//    float dx,dy,dz;

    float dx,dy,dz,dxs,dys,dzs,hs;
    int id,ii,il,it,iv;
    int lagbeg,lagend,ilag;
    float band,dcazm,dcdec,dxy,vrh,vrhpr,vrt,vrtpr,h;
    int omni;

    dx  = d_x[i] - d_x[j];
    dy  = d_y[i] - d_y[j];
    dz  = d_z[i] - d_z[j];
    dxs = dx*dx;
    dys = dy*dy;
    dzs = dz*dz;
    hs  = dxs + dys + dzs;

    if(hs <= dismxs)
    {
        if(hs < 0.0) hs = 0.0;
        h   = sqrtf(hs);


    //
    // Determine which lag this is and skip if outside the defined distance
    // tolerance:
    //
        if(h<=EPSLON){
            lagbeg = 1;
            lagend = 1;
        }
        else{
            lagbeg = -1;
            lagend = -1;
            for(ilag=2;ilag<=nlag+2;ilag++){
                if(h>=(xlag*(float)(ilag-2)-xltol) && h<=(xlag*(float)(ilag-2)+xltol)){
                    if(lagbeg<0) lagbeg = ilag;
                    lagend = ilag;
                }
            }

        }
        if(lagend>=0)
        {
        //			printf("dx=%f dy=%f dz=%fh=%f lagbeg=%d lagend=%d\n",dx,dy,dz,h,lagbeg,lagend);


        //
        // Definition of the direction corresponding to the current pair. All
        // directions are considered (overlapping of direction tolerance cones
        // is allowed):
        //
            for(id=0;id<ndir;id++){
            //
            // Check for an acceptable azimuth angle:
            //
                dxy = sqrtf(MAX((dxs+dys),0.0));
                if(dxy<EPSLON){
                    dcazm = 1.0;
                }
                else{
                    dcazm = (dx*d_uvxazm[id]+dy*d_uvyazm[id])/dxy;
                }
                if(fabsf(dcazm)>=d_csatol[id])
                {
            //
            // Check the horizontal bandwidth criteria (maximum deviation
            // perpendicular to the specified direction azimuth):
            //
                    band = d_uvxazm[id]*dy - d_uvyazm[id]*dx;
                    if(fabsf(band)<d_bandwh[id])
                    {
                        //fprintf(stdout,"dxy=%f\tdcazm=%f\tband=%f\n",dxy,dcazm,band);


                //
                // Check for an acceptable dip angle:
                //
                        if(dcazm<0.0) dxy = -dxy;
                        if(lagbeg==1)
                            dcdec = 0.0;
                        else{
                            dcdec = (dxy*d_uvhdec[id]+dz*d_uvzdec[id])/h;

                        }
                        band = d_uvhdec[id]*dz - d_uvzdec[id]*dxy;
                        if(fabsf(dcdec)>=d_csdtol[id] && fabsf(band)<=d_bandwd[id])
                        {
                    //
                    // Check the vertical bandwidth criteria (maximum deviation perpendicular
                    // to the specified dip direction):
                    //

                        //
                        // Check whether or not an omni-directional variogram is being computed:
                        //
                                omni = 0;
                                if(d_atol[id]>=90.0) omni = 1;
                        //
                        // This direction is acceptable - go ahead and compute all variograms:
                        //

                            //printf("dxy=%f dcazm=%f uvxazm[0]=%f uvyazm[0]=%f band=%f dcdec=%f omni=%d csdtol[0]=%f\n",dxy,dcazm,uvxazm[0],uvyazm[0],band,dcdec,omni,csdtol[0]);

                        //				fprintf(stdout,"dcazm=%f\tdcdec=%f\n",dcazm,dcdec);
                            for(iv=0;iv<nvarg;iv++){
                    //
                    // For this variogram, sort out which is the tail and the head value:
                    //
                                it = d_ivtype[iv];
                                if(dcazm>=0.0 && dcdec>=0.0){
                                    ii = d_ivtail[iv]-1;
                                    vrh   = d_vr[i+ii*(maxdat)];
                                    ii = d_ivhead[iv]-1;
                                    vrt   = d_vr[j+ii*(maxdat)];
                                    if(omni || it==2){
                                        ii    = d_ivhead[iv]-1;
                                        vrtpr = d_vr[i+ii*(maxdat)];
                                        ii    = d_ivtail[iv]-1;
                                        vrhpr = d_vr[j+ii*(maxdat)];
                                    }
                                }
                                else{
                                    ii = d_ivtail[iv]-1;
                                    vrh   = d_vr[j+ii*(maxdat)];
                                    ii = d_ivhead[iv]-1;
                                    vrt   = d_vr[i+ii*(maxdat)];
                                    if(omni || it==2){
                                        ii    = d_ivhead[iv]-1;
                                        vrtpr = d_vr[j+ii*(maxdat)];
                                        ii    = d_ivtail[iv]-1;
                                        vrhpr = d_vr[i+ii*(maxdat)];
                                    }
                                }
                    //
                    // Reject this pair on the basis of missing values:
                    //
                                if(vrt>=tmin && vrh>=tmin && vrt<=tmax && vrh<=tmax && it!=2 || (vrtpr>=tmin && vrhpr>=tmin && vrtpr<=tmax && vrhpr<=tmax))
                                {
                                    if(it==1 || it==5 || it>=9){
                                        for(il=lagbeg;il<=lagend;il++){
                                            ii = (id)*(nvarg)*((nlag)+2)+(iv)*((nlag)+2)+il -1;



                                            atomicAdd(&sh_np[ii],1.0);
                                            atomicAdd(&sh_dis[ii],(h));
                                            atomicAdd(&sh_tm[ii],(vrt));
                                            atomicAdd(&sh_hm[ii],(vrh));
                                            atomicAdd(&sh_gam[ii],((vrh-vrt)*(vrh-vrt)));

                                            if(omni){
                                                if(vrtpr>=tmin && vrhpr>=tmin && vrtpr<tmax && vrhpr<tmax){
                                                    atomicAdd(&sh_np[ii],1.0);
                                                    atomicAdd(&sh_dis[ii],(h));
                                                    atomicAdd(&sh_tm[ii],(vrtpr));
                                                    atomicAdd(&sh_hm[ii],(vrhpr));
                                                    atomicAdd(&sh_gam[ii],((vrhpr-vrtpr)*(vrhpr-vrtpr)));
                                    }
                                            }
                                        }
                                    }

                                    // The Traditional Cross Semivariogram:
                //
                                    else if(it==2){
                                        for(il=lagbeg;il<=lagend;il++){
                                            ii = (id)*(nvarg)*((nlag)+2)+(iv)*((nlag)+2)+il -1;
                                            atomicAdd(&sh_np[ii],1.0);
                                            atomicAdd(&sh_dis[ii],(h));
                                            atomicAdd(&sh_tm[ii],(0.5*(vrt+vrtpr)));
                                            atomicAdd(&sh_hm[ii],(0.5*(vrh+vrhpr)));
                                            atomicAdd(&sh_gam[ii],((vrhpr-vrh)*(vrt-vrtpr)));

                                        }
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}


__global__ void variogramKernel(    const int nd, const int irepo, const int maxdat, const int MAXVAR,
                                    float *d_x, float *d_y, float *d_z,
                                    const float EPSLON,
                                    const int nlag,
                                    const float xlag, const float xltol,
                                    const int mxdlv,
                                    DT *d_np, DT *d_dis, DT *d_gam, DT *d_hm, DT *d_tm,
                                    const float dismxs, const float tmax, const float tmin,
                                    const int ndir, const int nvarg,
                                    float *d_uvxazm,  float *d_uvyazm,  float *d_uvzdec,  float *d_uvhdec,
                                    float *d_csatol, float *d_csdtol, float *d_bandwh, float *d_bandwd,
                                    float *d_atol,
                                    int *d_ivtype, int *d_ivtail, int *d_ivhead,
                                    float *d_vr){

    int tidx=threadIdx.x;
    int tidy=threadIdx.y;
    int bidx=blockIdx.x;
    int bidy=blockIdx.y;
    int bdimx=blockDim.x;
    int bdimy=blockDim.y;
    int idx = bidx*bdimx + tidx;
    int idy = bidy*bdimy + tidy;
    int threadId = tidx + bdimx*tidy;
    int half_nd = nd/2;

    int i,j;
    extern __shared__ float buffer[];
    float *sh_np = &buffer[0];
    float *sh_dis = &buffer[mxdlv];
    float *sh_gam = &buffer[2*mxdlv];
    float *sh_hm = &buffer[3*mxdlv];
    float *sh_tm = &buffer[4*mxdlv];

    if (threadId < mxdlv){
        sh_np[threadId] = 0.0;
        sh_dis[threadId] = 0.0;
        sh_gam[threadId] = 0.0;
        sh_hm[threadId] = 0.0;
        sh_tm[threadId] = 0.0;
    }
    __syncthreads();

    if (idx < half_nd && idy < half_nd){

        j = idx + half_nd;
        i = idy;
        computeVariogram(i,j,nd,irepo,maxdat,MAXVAR,
            d_x,d_y,d_z,
            EPSLON,nlag,xlag,xltol,
            mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
            dismxs,tmax,tmin,ndir,nvarg,
            d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
            d_csatol, d_csdtol, d_bandwh, d_bandwd,
            d_atol,
            d_ivtype, d_ivtail, d_ivhead,
            d_vr);

        if (idx > idy){
            i = idy;
            j = idx;

            computeVariogram(i,j,nd,irepo,maxdat,MAXVAR,
                d_x,d_y,d_z,
                EPSLON,nlag,xlag,xltol,
                mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
                dismxs,tmax,tmin,ndir,nvarg,
                d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                d_csatol, d_csdtol, d_bandwh, d_bandwd,
                d_atol,
                d_ivtype, d_ivtail, d_ivhead,
                d_vr);

        } else if (idx < idy){
            i = idx + half_nd;
            j = idy + half_nd;

            computeVariogram(i,j,nd,irepo,maxdat,MAXVAR,
                d_x,d_y,d_z,
                EPSLON,nlag,xlag,xltol,
                mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
                dismxs,tmax,tmin,ndir,nvarg,
                d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                d_csatol, d_csdtol, d_bandwh, d_bandwd,
                d_atol,
                d_ivtype, d_ivtail, d_ivhead,
                d_vr);
        }

        if (idx == 0){
            i = idy;
            j = idy;

            computeVariogram(i,j,nd,irepo,maxdat,MAXVAR,
                d_x,d_y,d_z,
                EPSLON,nlag,xlag,xltol,
                mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
                dismxs,tmax,tmin,ndir,nvarg,
                d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                d_csatol, d_csdtol, d_bandwh, d_bandwd,
                d_atol,
                d_ivtype, d_ivtail, d_ivhead,
                d_vr);

        } else if (idy == 0){
            i = idx + half_nd;
            j = idx + half_nd;

            computeVariogram(i,j,nd,irepo,maxdat,MAXVAR,
                d_x,d_y,d_z,
                EPSLON,nlag,xlag,xltol,
                mxdlv,sh_np,sh_dis,sh_tm,sh_hm,sh_gam,
                dismxs,tmax,tmin,ndir,nvarg,
                d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                d_csatol, d_csdtol, d_bandwh, d_bandwd,
                d_atol,
                d_ivtype, d_ivtail, d_ivhead,
                d_vr);
        }


        __syncthreads();

        if (threadId < mxdlv){

            atomicAdd(&d_np[threadId],sh_np[threadId]);
            atomicAdd(&d_dis[threadId],sh_dis[threadId]);
            atomicAdd(&d_tm[threadId],sh_tm[threadId]);
            atomicAdd(&d_hm[threadId],sh_hm[threadId]);
            atomicAdd(&d_gam[threadId],sh_gam[threadId]);
        }
    }
}

extern "C" int gamvCUDA(
//      integer nd,irepo,maxdat,MAXVAR
	int *nd, int *irepo, int *maxdat, int *MAXVAR,
//      real x(maxdat),y(maxdat),z(maxdat)
	float **x, float **y, float **z,
//      real EPSLON
	float *EPSLON,
//      integer nlag
	int *nlag,
//      real xlag,xltol
	float *xlag, float *xltol,
//      integer mxdlv
	int *mxdlv,
//      real*8 np(mxdlv),dis(mxdlv),gam(mxdlv),hm(mxdlv),
//     + tm(mxdlv),hv(mxdlv),tv(mxdlv)
	double **np, double **dis, double **gam, double **hm, double **tm, double **hv, double **tv,
//      integer numThreads
	int *numThreads,
//      real*8 reducedVariables(7,mxdlv,numThreads)
	double *reducedVariables,
//      real dismxs,tmax,tmin
	float *dismxs, float *tmax, float *tmin,
//      integer ndir,nvarg
	int *ndir, int *nvarg,
//      real uvxazm(100),uvyazm(100),uvzdec(100),uvhdec(100)
	float **uvxazm, float **uvyazm, float **uvzdec, float **uvhdec,
//      real csatol(100),csdtol(100),bandwh(ndir),bandwd(ndir)
	float **csatol, float **csdtol, float **bandwh, float **bandwd,
//      real atol(ndir)
	float **atol,
//      integer ivtype(nvarg),ivtail(nvarg),ivhead(nvarg)
	int **ivtype, int **ivtail, int **ivhead,
//      real vr(maxdat,MAXVAR)
	float **vr
	)
{
	float *d_x,*d_y,*d_z;
	DT *d_np,*d_dis,*d_gam,*d_hm,*d_tm;
	DT *h_np,*h_dis,*h_gam,*h_hm,*h_tm;
	float *d_uvxazm,*d_uvyazm,*d_uvzdec,*d_uvhdec,*d_csatol,*d_csdtol,*d_bandwh,*d_bandwd,*d_atol,*d_vr;
	int *d_ivtype,*d_ivtail,*d_ivhead;
    hipSetDevice(2);
    dim3 threads(16,16,1);
//	dim3 blocks( (*maxdat/2 + threads.x - 1)/threads.x,(*maxdat/2 + threads.y - 1)/threads.y,1 );
	dim3 blocks( (*maxdat/2 + threads.x - 1)/threads.x,(*maxdat/2 + threads.y - 1)/threads.y,1 );

	h_np = (DT*)malloc(sizeof(DT)* *mxdlv);
	h_dis = (DT*)malloc(sizeof(DT)* *mxdlv);
	h_gam = (DT*)malloc(sizeof(DT)* *mxdlv);
	h_hm = (DT*)malloc(sizeof(DT)* *mxdlv);
	h_tm = (DT*)malloc(sizeof(DT)* *mxdlv);
	int shared_mem_size = sizeof(DT)*(*mxdlv*5);
    int i;
    for (i = 0; i < *mxdlv; i++){
        h_np[i] = 0.0;
        h_dis[i] = 0.0;
        h_gam[i] = 0.0;
        h_hm[i] = 0.0;
        h_tm[i] = 0.0;
    }
    hipMalloc( (void **)&d_x, sizeof(float) * (*maxdat) );
    //Check_CUDA_Error("malloc coord");
    hipMalloc( (void **)&d_y, sizeof(float) * (*maxdat) );
    //Check_CUDA_Error("malloc coord");
    hipMalloc( (void **)&d_z, sizeof(float) * (*maxdat) );
    //Check_CUDA_Error("malloc coord");
    hipMalloc( (void **)&d_np, sizeof(DT) * (*mxdlv) );
    //Check_CUDA_Error("malloc np, dis, gam, hm, tm");
    hipMalloc( (void **)&d_dis, sizeof(DT) * (*mxdlv) );
    //Check_CUDA_Error("malloc np, dis, gam, hm, tm");
    hipMalloc( (void **)&d_gam, sizeof(DT) * (*mxdlv) );
    //Check_CUDA_Error("malloc np, dis, gam, hm, tm");
    hipMalloc( (void **)&d_hm, sizeof(DT) * (*mxdlv) );
    //Check_CUDA_Error("malloc np, dis, gam, hm, tm");
    hipMalloc( (void **)&d_tm, sizeof(DT) * (*mxdlv) );
    //Check_CUDA_Error("malloc np, dis, gam, hm, tm");
    hipMalloc( (void **)&d_uvxazm, sizeof(float) * (100) );
    //Check_CUDA_Error("small mallocs ");
    hipMalloc( (void **)&d_uvyazm, sizeof(float) * (100) );
    //Check_CUDA_Error("small mallocs ");
    hipMalloc( (void **)&d_uvzdec, sizeof(float) * (100) );
    //Check_CUDA_Error("small mallocs ");
    hipMalloc( (void **)&d_uvhdec, sizeof(float) * (100) );
    //Check_CUDA_Error("small mallocs ");
    hipMalloc( (void **)&d_csatol, sizeof(float) * (100) );
    //Check_CUDA_Error("small mallocs ");
    hipMalloc( (void **)&d_csdtol, sizeof(float) * (100) );
    //Check_CUDA_Error("small mallocs ");
    hipMalloc( (void **)&d_bandwh, sizeof(float) * (*ndir) );
    //Check_CUDA_Error("small mallocs ");
    hipMalloc( (void **)&d_bandwd, sizeof(float) * (*ndir) );
    //Check_CUDA_Error("small mallocs ");
    hipMalloc( (void **)&d_atol, sizeof(float) * (*ndir) );
    //Check_CUDA_Error("small mallocs ");
    hipMalloc( (void **)&d_vr, sizeof(float) * (*maxdat* *MAXVAR) );
    //Check_CUDA_Error("small mallocs ");
    hipMalloc( (void **)&d_ivtype, sizeof(float) * (*nvarg) );
    //Check_CUDA_Error("iv mallocs");
    hipMalloc( (void **)&d_ivtail, sizeof(float) * (*nvarg) );
    //Check_CUDA_Error("iv mallocs");
    hipMalloc( (void **)&d_ivhead, sizeof(float) * (*nvarg) );
    //Check_CUDA_Error("iv mallocs");
    hipMemcpy( d_x, *x,sizeof(float) * (*maxdat), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy coords h -> d");
    hipMemcpy( d_y, *y,sizeof(float) * (*maxdat), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy coords h -> d");
    hipMemcpy( d_z, *z,sizeof(float) * (*maxdat), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy coords h -> d");
    hipMemcpy( d_np, h_np,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
    hipMemcpy( d_dis, h_dis,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
    hipMemcpy( d_gam, h_gam,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
    hipMemcpy( d_hm, h_hm,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
    hipMemcpy( d_tm, h_tm,sizeof(DT) * (*mxdlv), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy np, dis, gam, hm, tm h -> d");
    hipMemcpy( d_uvxazm, *uvxazm,sizeof(float) * (100), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy small data h -> d");
    hipMemcpy( d_uvyazm, *uvyazm,sizeof(float) * (100), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy small data h -> d");
    hipMemcpy( d_uvzdec, *uvzdec,sizeof(float) * (100), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy small data h -> d");
    hipMemcpy( d_uvhdec, *uvhdec,sizeof(float) * (100), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy small data h -> d");
    hipMemcpy( d_csatol, *csatol,sizeof(float) * (100), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy small data h -> d");
    hipMemcpy( d_csdtol, *csdtol,sizeof(float) * (100), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy small data h -> d");
    hipMemcpy( d_bandwh, *bandwh,sizeof(float) * (*ndir), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy small data h -> d");
    hipMemcpy( d_bandwd, *bandwd,sizeof(float) * (*ndir), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy small data h -> d");
    hipMemcpy( d_atol, *atol,sizeof(float) * (*ndir), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy small data h -> d");
    hipMemcpy( d_vr, *vr,sizeof(float) * (*maxdat* *MAXVAR), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy small data h -> d");
    hipMemcpy( d_ivtype, *ivtype,sizeof(float) * (*nvarg), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy iv var h -> d");
    hipMemcpy( d_ivtail, *ivtail,sizeof(float) * (*nvarg), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy iv var h -> d");
    hipMemcpy( d_ivhead, *ivhead,sizeof(float) * (*nvarg), hipMemcpyHostToDevice );
    //Check_CUDA_Error("cpy iv var h -> d");

/*
    printf("\n---------------PARAMS---------\n");
    printf("mxdlv %d, ndir %d, nvarg %d\n",*mxdlv,*ndir,*nvarg);
    printf("maxdat %d, MAXVAR %d, maxdat*MAXVAR %d\n", *maxdat, *MAXVAR,*maxdat* *MAXVAR);
    printf("------------------------------\n");
*/

/*	
    cudaEvent_t start, stop;
    float time;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);
    cudaEventRecord(start, 0);
*/	
    variogramKernel<<< blocks, threads,shared_mem_size >>>(*nd,*irepo,*maxdat,*MAXVAR,
                                        d_x,d_y,d_z,
                                        *EPSLON,
                                        *nlag,
                                        *xlag,*xltol,
                                        *mxdlv,
                                        d_np,d_dis,d_gam,d_hm,d_tm,
                                        *dismxs,*tmax,*tmin,
                                        *ndir,*nvarg,
                                        d_uvxazm,d_uvyazm,d_uvzdec,d_uvhdec,
                                        d_csatol,d_csdtol,d_bandwh,d_bandwd,
                                        d_atol,
                                        d_ivtype,d_ivtail,d_ivhead,
                                        d_vr);

/*	
    cudaDeviceSynchronize();
    Check_CUDA_Error("fitness kernel");
    cudaEventRecord(stop, 0);
    cudaEventSynchronize(stop);
    cudaEventElapsedTime(&time, start, stop);
    //        printf ("Time for the Optimized kernel: %f ms\n", time);
    printf ("GPU time: %f\n", time/1000);
    printf("------------------------------\n");
*/	
    hipMemcpy( h_np, d_np,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost);
    //Check_CUDA_Error("cpy d -> h");
    hipMemcpy( h_dis, d_dis,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost);
    //Check_CUDA_Error("cpy d -> h");
    hipMemcpy( h_gam, d_gam,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost);
    //Check_CUDA_Error("cpy d -> h");
    hipMemcpy( h_hm, d_hm,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost);
    //Check_CUDA_Error("cpy d -> h");
    hipMemcpy( h_tm, d_tm,sizeof(DT) * (*mxdlv),hipMemcpyDeviceToHost);
    //Check_CUDA_Error("cpy d -> h");

   // printf("np, dis, gam, hm, tm\n");
//    float sum_np = 0.0;
    for (i = 0; i < *mxdlv; i++){
        (*np)[i] = (double)h_np[i];
        (*dis)[i] = (double)h_dis[i];
        (*gam)[i] = (double)h_gam[i];
        (*hm)[i] = (double)h_hm[i];
        (*tm)[i] = (double)h_tm[i];
      //  printf("%lf\t, %lf\t, %lf\t, %lf\t, %lf\n",np[i],dis[i],gam[i],hm[i],tm[i]);
    }


    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    hipFree(d_np);
    hipFree(d_dis);
    hipFree(d_gam);
    hipFree(d_hm);
    hipFree(d_tm);
    hipFree(d_uvxazm);
    hipFree(d_uvyazm);
    hipFree(d_uvzdec);
    hipFree(d_uvhdec);
    hipFree(d_csatol);
    hipFree(d_csdtol);
    hipFree(d_bandwh);
    hipFree(d_bandwd);
    hipFree(d_atol);
    hipFree(d_vr);
    hipFree(d_ivtype);
    hipFree(d_ivtail);
    hipFree(d_ivhead);
    free(h_np);
    free(h_dis);
    free(h_gam);
    free(h_hm);
    free(h_tm);
	return 0;
//end routine
}

